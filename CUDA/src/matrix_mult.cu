#include "hip/hip_runtime.h"
// add.cu

#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA kernel function for GEMM operation
__global__ void gemm(int *a, int *b, int *c, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

extern "C" void add_arrays(int *a, int *b, int *c, int n) {
    int *d_a, *d_b, *d_c;
    size_t size = n * sizeof(int);

    // Allocate memory on the device (GPU)
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // Copy input arrays from host memory to device memory
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // Launch add() kernel on the GPU with one thread for each element
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    add<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, n);

    // Copy result array from device memory to host memory
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}
